#include "hip/hip_runtime.h"
#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <bit>
#include <bitset>
#include <bits/stdc++.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#define NUM_THREADS 256

using namespace std;

// generate 3 order combinations
void generate_combinations(vector<int> &combinations, int snp_size)
{
    for (int i = 0; i < snp_size - 2; i++)
    {
        for (int j = i + 1; j < snp_size - 1; j++)
        {
            for (int k = j + 1; k < snp_size; k++)
            {
                combinations.push_back(i);
                combinations.push_back(j);
                combinations.push_back(k);
            }
        }
    }
}

// build the bit table for the dataset
void build_bit_table(vector<vector<char>> &data, vector<vector<vector<bitset<64>>>> &bit_table, int size, int snp_size)
{
    for (int i = 0; i < snp_size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            int x = j / 64;
            int y = j % 64;
            bit_table[i][data[j][i] - '0'][x][y] = 1;
        }
    }
}

// construct the bin count
__global__ void build_contingency_table(uint64_t *bit_table, int *contingency_table, int *combinations, int num_combinations)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < num_combinations; i += stride)
    {
        int snp0 = combinations[i * 3 + 0];
        int snp1 = combinations[i * 3 + 1];
        int snp2 = combinations[i * 3 + 2];
        for (int idx = 0; idx < 27; idx++)
        {
            int snp0_type = idx / 9;
            int snp1_type = (idx % 9) / 3;
            int snp2_type = idx % 3;
            uint64_t t0 = bit_table[snp0 * 3 + snp0_type];
            uint64_t t1 = bit_table[snp1 * 3 + snp1_type];
            uint64_t t2 = bit_table[snp2 * 3 + snp2_type];
            uint64_t t = (t0 & t1 & t2);
            contingency_table[i * 27 + idx] += __popcll(t);
        }
    }
}

__global__ void k2_score(int *control_contingency_table, int *case_contingency_table, double *scores, int num_combinations)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < num_combinations; i += stride)
    {
        for (int idx = 0; idx < 27; idx++)
        {
            int case_count = case_contingency_table[i * 27 + idx];
            int control_count = control_contingency_table[i * 27 + idx];
            int total_count = case_count + control_count;
            double first_log = 0, second_log = 0;
            for (int b = 1; b <= total_count + 1; b++)
            {
                first_log += logf(b);
            }
            for (int d = 1; d <= case_count; d++)
            {
                second_log += logf(d);
            }
            for (int d = 1; d <= control_count; d++)
            {
                second_log += logf(d);
            }
            scores[i] += (first_log - second_log);
        }
    }
}

int main(int argc, char *argv[])
{

    // cuda variable setup
    int devId;
    hipGetDevice(&devId);
    int numSM;
    hipDeviceGetAttribute(&numSM, hipDeviceAttributeMultiprocessorCount, devId);
    int device_blks = 32 * numSM;

    int control_size = 0;
    int case_size = 0;
    int control_64_multiples = 0;
    int case_64_multiples = 0;
    int snp_size = 0;
    int num_combinations = 0;

    // read the dataset
    fstream fin;
    fin.open(argv[1], ios::in);
    string line, word;
    vector<char> sample;

    // (number of control samples) * snp_size
    vector<vector<char>> control_data;
    // (number of case samples ) * snp_size
    vector<vector<char>> case_data;

    while (getline(fin, line, '\n'))
    {
        istringstream s(line);
        while (std::getline(s, word, ','))
        {
            if (word == "X")
            {
                break;
            }
            sample.push_back(word[0]);
        }
        if (sample.size() > 0)
        {
            if (sample[sample.size() - 1] == '0')
            {
                sample.pop_back();
                control_data.push_back(sample);
                control_size++;
            }
            else
            {
                sample.pop_back();
                case_data.push_back(sample);
                case_size++;
            }
            sample.clear();
        }
    }
    fin.close();
    auto start_time = std::chrono::steady_clock::now();

    snp_size = control_data[0].size();                    // get the number of snps
    control_64_multiples = ceil(control_size * 1.0 / 64); // get the number of 64 multiples in control sample
    case_64_multiples = ceil(case_size * 1.0 / 64);       // get the number of 64 multiples in case sample

    // generate 3 order combinations
    vector<int> combinations;
    generate_combinations(combinations, snp_size);
    num_combinations = combinations.size() / 3;

    // initialize the bit table
    // dimension: snp_size * 3 * (number of 64 multiple in the sample (ceiling))
    vector<vector<vector<bitset<64>>>> control_bit_table(snp_size, vector<vector<bitset<64>>>(3, vector<bitset<64>>(control_64_multiples, 0)));
    vector<vector<vector<bitset<64>>>> case_bit_table(snp_size, vector<vector<bitset<64>>>(3, vector<bitset<64>>(case_64_multiples, 0)));

    // build the bit table
    build_bit_table(control_data, control_bit_table, control_size, snp_size);
    build_bit_table(case_data, case_bit_table, case_size, snp_size);

    control_data.clear();
    case_data.clear();

    // CUDA Implementation

    // Common
    int *d_combinations;                        // device copy
    int *local_combinations = &combinations[0]; // local_copy
    int control_bit_table_size = snp_size * 3 * control_64_multiples * sizeof(uint64_t);
    uint64_t *d_control_bit_table;                                                 // device copy of control bit table
    uint64_t *long_control_bit_table = (uint64_t *)malloc(control_bit_table_size); // host copy of control bit table
    for (int k = 0; k < control_64_multiples; k++)
    {
        for (int i = 0; i < snp_size; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                int index = k * snp_size * 3 + i * 3 + j;
                long_control_bit_table[index] = control_bit_table[i][j][k].to_ullong();
            }
        }
    }
    int control_contingency_table_size = num_combinations * 27 * sizeof(int);
    int *d_control_contingency_table; // device copy
    int *control_contingency_table = (int *)malloc(control_contingency_table_size);

    // int chunk = 64;
    hipMalloc((void **)&d_combinations, num_combinations * 3 * sizeof(int));
    hipMemcpy(d_combinations, local_combinations, num_combinations * 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_control_contingency_table, num_combinations * 27 * sizeof(int));
    hipMalloc((void **)&d_control_bit_table, snp_size * 3 * sizeof(uint64_t)); // allocate memory for control bit table device copy
                                                                                // hipMemset(d_control_bit_table, 0, chunk * 27 * sizeof(int));
    for (int sample_chunk = 0; sample_chunk < control_64_multiples; sample_chunk++)
    {
        hipMemcpy(d_control_bit_table, long_control_bit_table + sample_chunk * snp_size * 3 * sizeof(uint64_t), snp_size * 3 * sizeof(uint64_t), hipMemcpyHostToDevice);
        build_contingency_table<<<device_blks, NUM_THREADS>>>(d_control_bit_table, d_control_contingency_table, d_combinations, num_combinations);
    }
    hipFree(d_control_bit_table);
    hipFree(d_control_contingency_table);
    free(long_control_bit_table);
    control_bit_table.clear();

    ///////////////////////////////////////////////////////////////////////////////////

    // Case Part
    int case_bit_table_size = snp_size * 3 * case_64_multiples * sizeof(uint64_t);
    uint64_t *d_case_bit_table;                                              // device copy of case bit table
    uint64_t *long_case_bit_table = (uint64_t *)malloc(case_bit_table_size); // host copy of case bit table
    for (int k = 0; k < case_64_multiples; k++)
    {
        for (int i = 0; i < snp_size; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                int index = k * snp_size * 3 + i * 3 + j;
                long_case_bit_table[index] = case_bit_table[i][j][k].to_ullong();
            }
        }
    }

    // hipMalloc((void **)&d_case_bit_table, case_bit_table_size);                                    // allocate memory for case bit table device copy
    // hipMemcpy(d_case_bit_table, long_case_bit_table, case_bit_table_size, hipMemcpyHostToDevice); // copy case bit table memory from host to device
    int case_contingency_table_size = num_combinations * 27 * sizeof(int);
    int *d_case_contingency_table; // device copy
    int *case_contingency_table = (int *)malloc(case_contingency_table_size);

    hipMalloc((void **)&d_case_contingency_table, num_combinations * 27 * sizeof(int));
    hipMalloc((void **)&d_case_bit_table, snp_size * 3 * sizeof(uint64_t));
    for (int sample_chunk = 0; sample_chunk < case_64_multiples; sample_chunk++)
    {
        hipMemcpy(d_case_bit_table, long_case_bit_table + sample_chunk * snp_size * 3 * sizeof(uint64_t), snp_size * 3 * sizeof(uint64_t), hipMemcpyHostToDevice);
        build_contingency_table<<<device_blks, NUM_THREADS>>>(d_case_bit_table, d_case_contingency_table, d_combinations, num_combinations);
    }
    hipFree(d_case_bit_table);
    hipFree(d_case_contingency_table);
    hipFree(d_combinations);
    free(long_case_bit_table);
    case_bit_table.clear();

    // calculate the k2 score and return the score and resulting combination
    double *scores = (double *)malloc(sizeof(double) * num_combinations); // host copy
    double *d_scores;                                                     // device copy
    //
    // hipMemset(d_scores, 0, sizeof(double) * num_combinations);           // set scores device memory
    hipMalloc((void **)&d_scores, sizeof(double) * num_combinations); // allocate scores device memory
    hipMalloc((void **)&d_control_contingency_table, num_combinations * 27 * sizeof(int));
    hipMalloc((void **)&d_case_contingency_table, num_combinations * 27 * sizeof(int));
    // for (int start_idx = 0; start_idx < num_combinations; start_idx += chunk)
    //{
    //     int chunk_size = min(chunk, (num_combinations - start_idx));
    hipMemcpy(d_control_contingency_table, control_contingency_table, num_combinations * 27 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_case_contingency_table, case_contingency_table, num_combinations * 27 * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_scores, 0, num_combinations * sizeof(double));
    k2_score<<<device_blks, NUM_THREADS>>>(d_control_contingency_table, d_case_contingency_table, d_scores, num_combinations);
    hipMemcpy(scores, d_scores, sizeof(double) * num_combinations, hipMemcpyDeviceToHost);
    //}
    hipFree(d_scores);
    hipFree(d_control_contingency_table);
    hipFree(d_case_contingency_table);

    int best_idx = 0;
    for (int i = 0; i < num_combinations; i++)
    {
        // cout << scores[i] << endl;
        best_idx = (scores[i] < scores[best_idx]) ? i : best_idx;
    }
    cout << "The lowest K2 score: " << scores[best_idx] << endl;
    cout << "The most likely combination of snps: " << combinations[best_idx * 3 + 0] << " " << combinations[best_idx * 3 + 1] << " " << combinations[best_idx * 3 + 2] << endl;
    free(scores);
    auto end_time = std::chrono::steady_clock::now();

    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    cout << "Finish in " << seconds << " seconds." << endl;

    return 0;
}