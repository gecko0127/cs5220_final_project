#include "hip/hip_runtime.h"
#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <bit>
#include <bitset>
#include <bits/stdc++.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#define NUM_THREADS 256

using namespace std;

void generate_combinations(uint16_t *combinations, int snp_size)
{
    uint16_t i, j, k;
    int count = 0;
    for (i = 0; i < snp_size - 2; i++)
    {
        for (j = i + 1; j < snp_size - 1; j++)
        {
            for (k = j + 1; k < snp_size; k++)
            {
                combinations[count++] = i;
                combinations[count++] = j;
                combinations[count++] = k;
            }
        }
    }
}

void build_bit_table(vector<vector<char>> &data, uint64_t *bit_table, int size, int snp_size)
{
    uint8_t multiples = ceil(size * 1.0 / 64);
    for (int i = 0; i < snp_size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            int x = j / 64;
            int y = j % 64;
            bit_table[i * 3 * multiples + (data[j][i] - '0') * multiples + x] |= ((uint64_t)1 << y);
        }
    }
}

__global__ void build_contingency_table(uint64_t *bit_table, uint16_t *contingency_table, uint16_t *combinations, int num_sample_64_mutiples, int num_combinations)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < num_combinations; i += blockDim.x * gridDim.x)
    {
        uint8_t idx, j;
        uint64_t t;
        for (idx = 0; idx < 27; idx++)
        {
            for (j = 0; j < num_sample_64_mutiples; j++)
            {
                t = ((bit_table[combinations[i * 3 + 0] * 3 * num_sample_64_mutiples + (idx / 9) * num_sample_64_mutiples + j]) & (bit_table[combinations[i * 3 + 1] * 3 * num_sample_64_mutiples + ((idx % 9) / 3) * num_sample_64_mutiples + j]) & (bit_table[combinations[i * 3 + 2] * 3 * num_sample_64_mutiples + (idx % 3) * num_sample_64_mutiples + j]));
                contingency_table[i * 27 + idx] += __popcll(t);
            }
        }
    }
}

__global__ void k2_score(uint16_t *control_contingency_table, uint16_t *case_contingency_table, float *scores, int num_combinations)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < num_combinations; i += blockDim.x * gridDim.x)
    {
        for (int idx = 0; idx < 27; idx++)
        {
            for (int b = 1; b <= case_contingency_table[i * 27 + idx] + control_contingency_table[i * 27 + idx] + 1; b++)
            {
                scores[i] += logf(b);
            }
            for (int d = 1; d <= case_contingency_table[i * 27 + idx]; d++)
            {
                scores[i] -= logf(d);
            }
            for (int d = 1; d <= control_contingency_table[i * 27 + idx]; d++)
            {
                scores[i] -= logf(d);
            }
        }
    }
}

int main(int argc, char *argv[])
{

    // cuda variable setup
    int devId;
    hipGetDevice(&devId);
    int numSM;
    hipDeviceGetAttribute(&numSM, hipDeviceAttributeMultiprocessorCount, devId);
    int device_blks = 32 * numSM;
    uint16_t control_size = 0, case_size = 0;
    uint16_t snp_size = 0;

    vector<vector<char>> control_data;
    vector<vector<char>> case_data;
    fstream fin;
    fin.open(argv[1], ios::in);
    string line, word;
    vector<char> sample;
    while (getline(fin, line, '\n'))
    {
        istringstream s(line);
        while (std::getline(s, word, ','))
        {
            if (word == "X")
            {
                break;
            }
            sample.push_back(word[0]);
        }
        if (sample.size() > 0)
        {
            if (sample[sample.size() - 1] == '0')
            {
                sample.pop_back();
                control_data.push_back(sample);
                control_size++;
            }
            else
            {
                sample.pop_back();
                case_data.push_back(sample);
                case_size++;
            }
            sample.clear();
        }
    }
    fin.close();
    auto start_time = std::chrono::steady_clock::now();
    snp_size = control_data[0].size();

    uint8_t control_64_multiples = ceil(control_size * 1.0 / 64), case_64_multiples = ceil(case_size * 1.0 / 64);
    int num_combinations = snp_size * (snp_size - 1) * (snp_size - 2) / 6;
    uint64_t *control_bit_table = (uint64_t *)malloc(snp_size * 3 * control_64_multiples * sizeof(uint64_t));
    memset(control_bit_table, 0, snp_size * 3 * control_64_multiples * sizeof(uint64_t));
    uint64_t *case_bit_table = (uint64_t *)malloc(snp_size * 3 * case_64_multiples * sizeof(uint64_t));
    memset(case_bit_table, 0, snp_size * 3 * case_64_multiples * sizeof(uint64_t));

    build_bit_table(control_data, control_bit_table, control_size, snp_size);
    build_bit_table(case_data, case_bit_table, case_size, snp_size);

    uint16_t *combinations = (uint16_t *)malloc(num_combinations * 3 * sizeof(uint16_t));
    generate_combinations(combinations, snp_size);
    uint16_t *d_combinations;
    hipMalloc((void **)&d_combinations, 3 * num_combinations * sizeof(uint16_t));
    hipMemcpy(d_combinations, combinations, 3 * num_combinations * sizeof(uint16_t), hipMemcpyHostToDevice);

    uint64_t *d_control_bit_table;
    hipMalloc((void **)&d_control_bit_table, snp_size * 3 * control_64_multiples * sizeof(uint64_t));
    hipMemcpy(d_control_bit_table, control_bit_table, snp_size * 3 * control_64_multiples * sizeof(uint64_t), hipMemcpyHostToDevice);
    free(control_bit_table);
    uint16_t *d_control_contingency_table;

    hipMalloc((void **)&d_control_contingency_table, num_combinations * 27 * sizeof(uint16_t));
    build_contingency_table<<<device_blks, NUM_THREADS>>>(d_control_bit_table, d_control_contingency_table, d_combinations, control_64_multiples, num_combinations);
    hipFree(d_control_bit_table);
    uint64_t *d_case_bit_table;
    hipMalloc((void **)&d_case_bit_table, snp_size * 3 * case_64_multiples * sizeof(uint64_t));
    hipMemcpy(d_case_bit_table, case_bit_table, snp_size * 3 * case_64_multiples * sizeof(uint64_t), hipMemcpyHostToDevice);
    free(case_bit_table);
    uint16_t *d_case_contingency_table;
    hipMalloc((void **)&d_case_contingency_table, num_combinations * 27 * sizeof(uint16_t));
    build_contingency_table<<<device_blks, NUM_THREADS>>>(d_case_bit_table, d_case_contingency_table, d_combinations, case_64_multiples, num_combinations);
    hipFree(d_case_bit_table);
    hipFree(d_combinations);

    float *d_scores;
    hipMalloc((void **)&d_scores, sizeof(float) * num_combinations);
    hipMemset(d_scores, 0, sizeof(float) * num_combinations);
    k2_score<<<device_blks, NUM_THREADS>>>(d_control_contingency_table, d_case_contingency_table, d_scores, num_combinations);
    hipFree(d_control_contingency_table);
    hipFree(d_case_contingency_table);
    float *scores = (float *)malloc(sizeof(float) * num_combinations);
    hipMemcpy(scores, d_scores, sizeof(float) * num_combinations, hipMemcpyDeviceToHost);
    hipFree(d_scores);
    int best_idx = 0;
    for (int i = 0; i < num_combinations; i++)
    {
        best_idx = (scores[i] < scores[best_idx]) ? i : best_idx;
    }
    cout << "The lowest K2 score: " << scores[best_idx] << endl;
    cout << "The most likely combination of snps: " << combinations[best_idx * 3 + 0] << " " << combinations[best_idx * 3 + 1] << " " << combinations[best_idx * 3 + 2] << endl;
    auto end_time = std::chrono::steady_clock::now();

    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    cout << "Finish in " << seconds << " seconds." << endl;

    free(scores);
    free(combinations);
    return 0;
}
