#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <bit>
#include <bitset>
#include <bits/stdc++.h>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

// generate 3 order combinations
void generate_combinations(vector<vector<char>> &combinations, int snp_size)
{
    for (int i = 0; i < snp_size - 2; i++)
    {
        for (int j = i + 1; j < snp_size - 1; j++)
        {
            for (int k = j + 1; k < snp_size; k++)
            {
                vector<char> combination = {char(i + '0'), char(j + '0'), char(k + '0')};
                combinations.push_back(combination);
            }
        }
    }
}

// build the bit table for the dataset
void build_bit_table(vector<vector<char>> &data, vector<vector<vector<bitset<64>>>> &bit_table, int size, int snp_size)
{
    for (int i = 0; i < snp_size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            int x = j / 64;
            int y = j % 64;
            bit_table[i][data[j][i] - '0'][x][y] = 1;
        }
    }
}

// build the contingency table from the bit table
void build_contingency_table(vector<vector<vector<bitset<64>>>> &bit_table, vector<vector<int>> &contingency_table, vector<vector<char>> &combinations, int size, int snp_size)
{
    for (int i = 0; i < combinations.size(); i++)
    {
        int snp0 = combinations[i][0] - '0';
        int snp1 = combinations[i][1] - '0';
        int snp2 = combinations[i][2] - '0';
        for (int idx = 0; idx < 27; idx++)
        {
            int snp0_type = idx / 9;
            int snp1_type = (idx % 9) / 3;
            int snp2_type = idx % 3;
            int count = 0;
            for (int i = 0; i < bit_table[snp0][0].size(); i++)
            {
                count += (bit_table[snp0][snp0_type][i] & bit_table[snp1][snp1_type][i] & bit_table[snp2][snp2_type][i]).count();
            }
            contingency_table[i][idx] = count;
        }
    }
}

// calculate k2 score
pair<vector<char>, double> k2_score(vector<vector<int>> &control_contingency_table, vector<vector<int>> &case_contingency_table, int snp_size, vector<vector<char>> &combinations)
{
    double k2 = DBL_MAX;
    vector<char> final_snp;
    for (int i = 0; i < combinations.size(); i++)
    {
        double score = 0;
        for (int idx = 0; idx < 27; idx++)
        {
            int case_count = case_contingency_table[i][idx];
            int control_count = control_contingency_table[i][idx];
            int total_count = case_count + control_count;
            double first_log = 0, second_log = 0;
            for (int b = 1; b <= total_count + 1; b++)
            {
                first_log += log(b);
            }
            for (int d = 1; d <= case_count; d++)
            {
                second_log += log(d);
            }
            for (int d = 1; d <= control_count; d++)
            {
                second_log += log(d);
            }
            score += (first_log - second_log);
        }

        if (score < k2)
        {
            k2 = score;
            final_snp = combinations[i];
        }
    }
    return {final_snp, k2};
}

__global__ void calculate_k2_score(int *d_case_table, int *d_control_table, double *d_scores, int num_combinations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_combinations) {
        double score = 0.0;
        for (int j = 0; j < 27; j++) {
            int case_count = d_case_table[idx * 27 + j];
            int control_count = d_control_table[idx * 27 + j];
            double first_log = 0.0, second_log = 0.0;
            for (int k = 1; k <= case_count + control_count; k++) {
                first_log += log((double)k);
            }
            for (int k = 1; k <= case_count; k++) {
                second_log += log((double)k);
            }
            for (int k = 1; k <= control_count; k++) {
                second_log += log((double)k);
            }
            score += (first_log - second_log);
        }
        d_scores[idx] = score;
    }
}

int main(int argc, char *argv[])
{
    int control_size = 0;
    int case_size = 0;
    int snp_size = 0;
    bool debug = false; // TODO: set to false if do not want to print out debug info

    // read the dataset
    fstream fin;
    fin.open(argv[1], ios::in);
    string line, word;
    vector<char> sample;

    // (number of control samples) * snp_size
    vector<vector<char>> control_data;

    // (number of case samples ) * snp_size
    vector<vector<char>> case_data;
    while (getline(fin, line, '\n'))
    {
        istringstream s(line);
        while (std::getline(s, word, ','))
        {
            if (word == "X")
            {
                break;
            }
            sample.push_back(word[0]);
        }
        if (sample.size() > 0)
        {
            if (sample[sample.size() - 1] == '0')
            {
                sample.pop_back();
                control_data.push_back(sample);
                control_size++;
            }
            else
            {
                sample.pop_back();
                case_data.push_back(sample);
                case_size++;
            }
            sample.clear();
        }
    }
    fin.close();
    // get the number of snps
    snp_size = control_data[0].size();

    if (debug)
    {
        cout << "first stage: read in data\n"
             << endl;

        cout << "this is control data: " << endl;
        for (int i = 0; i < control_size; i++)
        {
            cout << "first sample: ";
            for (int j = 0; j < snp_size; j++)
            {
                cout << control_data[i][j] << " ";
            }
            cout << endl;
        }

        cout << endl;

        cout << "this is case data: " << endl;
        for (int i = 0; i < case_size; i++)
        {
            cout << "first sample: ";
            for (int j = 0; j < snp_size; j++)
            {
                cout << case_data[i][j] << " ";
            }
            cout << endl;
        }

        cout << "-----------------------------------------------" << endl;
        cout << endl;
    }

    // generate 3 order combinations (each row is a combination)
    vector<vector<char>> combinations;
    generate_combinations(combinations, snp_size);

    if (debug)
    {
        cout << "second stage: generate the number of combinations of snps\n"
             << endl;
        for (auto &combination : combinations)
        {
            cout << combination[0] << " " << combination[1] << " " << combination[2] << endl;
        }
        cout << "\nthere are " << combinations.size() << " of combinations in total." << endl;

        cout << "-----------------------------------------------------" << endl;
    }

    // initialize the bit table
    // dimension: snp_size * 3 * (number of 64 multiple in the sample (ceiling))
    vector<vector<vector<bitset<64>>>> control_bit_table(snp_size, vector<vector<bitset<64>>>(3, vector<bitset<64>>(ceil(control_size * 1.0 / 64), 0)));
    vector<vector<vector<bitset<64>>>> case_bit_table(snp_size, vector<vector<bitset<64>>>(3, vector<bitset<64>>(ceil(case_size * 1.0 / 64), 0)));

    // build the bit table
    build_bit_table(control_data, control_bit_table, control_size, snp_size);
    build_bit_table(case_data, case_bit_table, case_size, snp_size);

    if (debug)
    {
        cout << "This is the third stage: building bit table\n"
             << endl;

        cout << "This is the control bit table: " << endl;
        for (int i = 0; i < snp_size; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                cout << "snp: " << i << "; genotype: " << j << " : ";
                for (auto &c : control_bit_table[i][j])
                {
                    cout << c << " ";
                }
                cout << endl;
            }
        }

        cout << endl;

        cout << "This is the case bit table: " << endl;
        for (int i = 0; i < snp_size; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                cout << "snp: " << i << "; genotype: " << j << " : ";
                for (auto &c : case_bit_table[i][j])
                {
                    cout << c << " ";
                }
                cout << endl;
            }
        }

        cout << "-------------------------------------------------------" << endl;
        cout << endl;
    }

    // initialize the contingency table
    // (number of combinations) * (number of genotype combinations: 3 * 3 * 3)
    vector<vector<int>> control_contingency_table(combinations.size(), vector<int>(27, 0));
    vector<vector<int>> case_contingency_table(combinations.size(), vector<int>(27, 0));

    // build the contingency table
    build_contingency_table(control_bit_table, control_contingency_table, combinations, control_size, snp_size);
    build_contingency_table(case_bit_table, case_contingency_table, combinations, case_size, snp_size);

    if (debug)
    {
        cout << "This is the fourth stage: building contingency table\n"
             << endl;

        cout << "This is the control contingency table: " << endl;
        for (int i = 0; i < combinations.size(); i++)
        {
            cout << "snp0: " << combinations[i][0] << "; snp1: " << combinations[i][1] << "; snp2: " << combinations[i][2] << endl;
            for (int idx = 0; idx < 27; idx++)
            {
                int snp0_type = idx / 9;
                int snp1_type = (idx % 9) / 3;
                int snp2_type = idx % 3;
                if (control_contingency_table[i][idx] != 0)
                {
                    cout << "genotype: " << snp0_type << " " << snp1_type << " " << snp2_type << " : " << control_contingency_table[i][idx] << endl;
                }
            }
            cout << endl;
        }

        cout << endl;

        cout << "This is the case contingency table: " << endl;
        for (int i = 0; i < combinations.size(); i++)
        {
            cout << "snp0: " << combinations[i][0] << "; snp1: " << combinations[i][1] << "; snp2: " << combinations[i][2] << endl;
            for (int idx = 0; idx < 27; idx++)
            {
                int snp0_type = idx / 9;
                int snp1_type = (idx % 9) / 3;
                int snp2_type = idx % 3;
                if (case_contingency_table[i][idx] != 0)
                {
                    cout << "genotype: " << snp0_type << " " << snp1_type << " " << snp2_type << " : " << case_contingency_table[i][idx] << endl;
                }
            }
            cout << endl;
        }
    }


    double *d_scores;
    int *d_case_table, *d_control_table;

    // Allocate memory on the device
    hipMalloc(&d_scores, combinations.size() * sizeof(double));
    hipMalloc(&d_case_table, combinations.size() * 27 * sizeof(int));
    hipMalloc(&d_control_table, combinations.size() * 27 * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_case_table, &case_contingency_table[0][0], combinations.size() * 27 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_control_table, &control_contingency_table[0][0], combinations.size() * 27 * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (combinations.size() + blockSize - 1) / blockSize;

    // Launch the kernel
    calculate_k2_score<<<numBlocks, blockSize>>>(d_case_table, d_control_table, d_scores, combinations.size());

    // Copy results back to the host
    vector<double> scores(combinations.size());
    hipMemcpy(&scores[0], d_scores, combinations.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Find the minimum score and corresponding combination
    double minScore = DBL_MAX;
    vector<char> bestCombination;
    for (int i = 0; i < scores.size(); i++) {
        if (scores[i] < minScore) {
            minScore = scores[i];
            bestCombination = combinations[i];
        }
    }

    // Free device memory
    hipFree(d_scores);
    hipFree(d_case_table);
    hipFree(d_control_table);

    cout << "The lowest K2 score: " << minScore << endl;
    cout << "The most likely combination of snps: " << bestCombination[0] << " " << bestCombination[1] << " " << bestCombination[2] << endl;

    return 0;
}